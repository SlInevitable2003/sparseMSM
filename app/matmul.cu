#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include "ec/bigint.hpp"
#include "ec/alt_bn128.hpp"
#include "ec_cuda/bigint.cuh"
#include "ec_cuda/alt_bn128.cuh"
using namespace std;

#define whatis(x) cerr << #x << " = " << x << endl
#define whatare(pr) cerr << #pr << " = (" << pr.first << ", " << pr.second << ")" << endl

#define SEED 0

Bit randBit() { return Bit(rand() & 1); }
Int randInt() { Int ret = 0; for (int i = 0; i < 3; i++) ret = (ret << 31) | rand(); return ret; }
template<size_t t> Bigint<t> randBigint() { Bigint<t> ret; for (int i = 0; i < t; i++) ret.data[i] = randInt(); return ret; }
alt_bn128_Fp rand_alt_bn128_Fp() 
{
    Bigint<4> ret = randBigint<4>();
    while (larger_or_eq(ret.data, alt_bn128::pri.data, 4)) ret = randBigint<4>();
    return {ret};
}

const int n = 1 << 4;
alt_bn128_Fp A[n][n], B[n][n], C[n][n], D[n][n]; // total 3 * n * n * 4 * 8B = 24KB

void param_setup()
{
    cuda::Bigint<4> *pri_dev, *R_square_dev;
    Int *pri_inv_dev;
    cuda::alt_bn128_Fp* one_dev, *zero_dev;
    hipMalloc((void **)&pri_dev, sizeof(cuda::Bigint<4>));
    hipMalloc((void **)&R_square_dev, sizeof(cuda::Bigint<4>));
    hipMalloc((void **)&pri_inv_dev, sizeof(Int));
    hipMalloc((void **)&one_dev, sizeof(cuda::alt_bn128_Fp));
    hipMalloc((void **)&zero_dev, sizeof(cuda::alt_bn128_Fp));

    hipMemcpy((void *)pri_dev, &alt_bn128::pri, sizeof(Bigint<4>), hipMemcpyHostToDevice);
    hipMemcpy((void *)R_square_dev, &alt_bn128::R_square, sizeof(Bigint<4>), hipMemcpyHostToDevice);
    hipMemcpy((void *)pri_inv_dev, &alt_bn128::pri_inv, sizeof(Int), hipMemcpyHostToDevice);
    hipMemcpy((void *)one_dev, &alt_bn128::one, sizeof(alt_bn128_Fp), hipMemcpyHostToDevice);
    hipMemcpy((void *)zero_dev, &alt_bn128::zero, sizeof(alt_bn128_Fp), hipMemcpyHostToDevice);

    cuda::alt_bn128::param_setup<<<1, 1>>>(pri_dev, R_square_dev, pri_inv_dev, one_dev, zero_dev);

    hipFree(pri_dev);
    hipFree(R_square_dev);
    hipFree(pri_inv_dev);
    hipFree(one_dev);
    hipFree(zero_dev);
}

__global__ void matmul(cuda::alt_bn128_Fp *A, cuda::alt_bn128_Fp *B, cuda::alt_bn128_Fp *D, const int n)
{
    for (int i = 0; i < n; i++) for (int j = 0; j < n; j++) {
        for (int k = 0; k < n; k++) D[i * n + j] = D[i * n + j] + (A[i * n + k] * B[k * n + j]);
        D[i * n + j].mont_unrepr();
    }
}

int main(int argc, char *argv[])
{
    for (int i = 0; i < n; i++) for (int j = 0; j < n; j++) {
        A[i][j] = rand_alt_bn128_Fp(), B[i][j] = rand_alt_bn128_Fp(), C[i][j] = alt_bn128::zero, D[i][j] = alt_bn128::zero;
        A[i][j].mont_repr(), B[i][j].mont_repr();
    }
    for (int i = 0; i < n; i++) for (int j = 0; j < n; j++) {
        for (int k = 0; k < n; k++) C[i][j] = C[i][j] + (A[i][k] * B[k][j]);
        C[i][j].mont_unrepr();
    }

    cuda::alt_bn128_Fp *dev_A, *dev_B, *dev_D;
    hipMalloc((void **)&dev_A, n * n * sizeof(alt_bn128_Fp));
    hipMalloc((void **)&dev_B, n * n * sizeof(alt_bn128_Fp));
    hipMalloc((void **)&dev_D, n * n * sizeof(alt_bn128_Fp));

    hipMemcpy((void *)dev_A, &A[0][0], n * n * sizeof(alt_bn128_Fp), hipMemcpyHostToDevice);
    hipMemcpy((void *)dev_B, &B[0][0], n * n * sizeof(alt_bn128_Fp), hipMemcpyHostToDevice);
    hipMemcpy((void *)dev_D, &D[0][0], n * n * sizeof(alt_bn128_Fp), hipMemcpyHostToDevice);

    matmul<<<1, 1>>>(dev_A, dev_B, dev_D, n);

    hipMemcpy(&D[0][0], (void *)dev_D, n * n * sizeof(alt_bn128_Fp), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) for (int j = 0; j < n; j++) D[i][j].print_hex();

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_D);

    return 0;
}