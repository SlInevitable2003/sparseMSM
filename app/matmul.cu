#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include "ec/bigint.hpp"
#include "ec/alt_bn128.hpp"
#include "ec_cuda/bigint.cuh"
#include "ec_cuda/alt_bn128.cuh"
using namespace std;

#define whatis(x) cerr << #x << " = " << x << endl
#define whatare(pr) cerr << #pr << " = (" << pr.first << ", " << pr.second << ")" << endl

#define SEED 0

Bit randBit() { return Bit(rand() & 1); }
Int randInt() { Int ret = 0; for (int i = 0; i < 3; i++) ret = (ret << 31) | rand(); return ret; }
template<size_t t> Bigint<t> randBigint() { Bigint<t> ret; for (int i = 0; i < t; i++) ret.data[i] = randInt(); return ret; }
alt_bn128_Fp rand_alt_bn128_Fp() 
{
    Bigint<4> ret = randBigint<4>();
    while (larger_or_eq(ret.data, alt_bn128::pri.data, 4)) ret = randBigint<4>();
    return {ret};
}

const int n = 1 << 4;
alt_bn128_Fp A[n][n], B[n][n], C[n][n], D[n][n]; // total 3 * n * n * 4 * 8B = 24KB

void param_setup()
{
    cuda::Bigint<4> *pri_dev, *R_square_dev;
    Int *pri_inv_dev;
    cuda::alt_bn128_Fp* one_dev, *zero_dev;
    hipMalloc((void **)&pri_dev, sizeof(cuda::Bigint<4>));
    hipMalloc((void **)&R_square_dev, sizeof(cuda::Bigint<4>));
    hipMalloc((void **)&pri_inv_dev, sizeof(Int));
    hipMalloc((void **)&one_dev, sizeof(cuda::alt_bn128_Fp));
    hipMalloc((void **)&zero_dev, sizeof(cuda::alt_bn128_Fp));

    hipMemcpy((void *)pri_dev, &alt_bn128::pri, sizeof(Bigint<4>), hipMemcpyHostToDevice);
    hipMemcpy((void *)R_square_dev, &alt_bn128::R_square, sizeof(Bigint<4>), hipMemcpyHostToDevice);
    hipMemcpy((void *)pri_inv_dev, &alt_bn128::pri_inv, sizeof(Int), hipMemcpyHostToDevice);
    hipMemcpy((void *)one_dev, &alt_bn128::one, sizeof(alt_bn128_Fp), hipMemcpyHostToDevice);
    hipMemcpy((void *)zero_dev, &alt_bn128::zero, sizeof(alt_bn128_Fp), hipMemcpyHostToDevice);

    cuda::alt_bn128::param_setup<<<1, 1>>>(pri_dev, R_square_dev, pri_inv_dev, one_dev, zero_dev);

    hipFree(pri_dev);
    hipFree(R_square_dev);
    hipFree(pri_inv_dev);
    hipFree(one_dev);
    hipFree(zero_dev);
}

#define BLK_SIZ (1 << 2)

__global__ void matmul(cuda::alt_bn128_Fp *A, cuda::alt_bn128_Fp *B, cuda::alt_bn128_Fp *D, const int n)
{
    __shared__ cuda::alt_bn128_Fp shareA[BLK_SIZ][BLK_SIZ];
    __shared__ cuda::alt_bn128_Fp shareB[BLK_SIZ][BLK_SIZ];
    __syncthreads();

    cuda::alt_bn128_Fp t = cuda::alt_bn128::zero;
    for (int i = 0; i < n / BLK_SIZ; i++) {
        shareA[threadIdx.x][threadIdx.y] = A[(blockIdx.x * BLK_SIZ + threadIdx.x) * n + (i * BLK_SIZ + threadIdx.y)];
        shareB[threadIdx.x][threadIdx.y] = B[(i * BLK_SIZ + threadIdx.x) * n + (blockIdx.y * BLK_SIZ + threadIdx.y)];
        __syncthreads();
        for (int j = 0; j < BLK_SIZ; j++) t = t + (shareA[threadIdx.x][j] * shareB[j][threadIdx.y]);
        __syncthreads();
    }

    t.mont_unrepr();
    D[(blockIdx.x * BLK_SIZ + threadIdx.x) * n + (blockIdx.y * BLK_SIZ + threadIdx.y)] = t;
}

// __global__ void matmul(cuda::alt_bn128_Fp *A, cuda::alt_bn128_Fp *B, cuda::alt_bn128_Fp *D, const int n)
// {
//     for (int i = 0; i < n; i++) for (int j = 0; j < n; j++) {
//         for (int k = 0; k < n; k++) D[i * n + j] = D[i * n + j] + (A[i * n + k] * B[k * n + j]);
//         D[i * n + j].mont_unrepr();
//     }
// }

int main(int argc, char *argv[])
{
    param_setup();

    for (int i = 0; i < n; i++) for (int j = 0; j < n; j++) {
        A[i][j] = rand_alt_bn128_Fp(), B[i][j] = rand_alt_bn128_Fp(), C[i][j] = alt_bn128::zero, D[i][j] = alt_bn128::zero;
        A[i][j].mont_repr(), B[i][j].mont_repr();
    }
    for (int i = 0; i < n; i++) for (int j = 0; j < n; j++) {
        for (int k = 0; k < n; k++) C[i][j] = C[i][j] + (A[i][k] * B[k][j]);
        C[i][j].mont_unrepr();
    }

    cuda::alt_bn128_Fp *dev_A, *dev_B, *dev_D;
    hipMalloc((void **)&dev_A, n * n * sizeof(alt_bn128_Fp));
    hipMalloc((void **)&dev_B, n * n * sizeof(alt_bn128_Fp));
    hipMalloc((void **)&dev_D, n * n * sizeof(alt_bn128_Fp));

    hipMemcpy((void *)dev_A, &A[0][0], n * n * sizeof(alt_bn128_Fp), hipMemcpyHostToDevice);
    hipMemcpy((void *)dev_B, &B[0][0], n * n * sizeof(alt_bn128_Fp), hipMemcpyHostToDevice);
    hipMemcpy((void *)dev_D, &D[0][0], n * n * sizeof(alt_bn128_Fp), hipMemcpyHostToDevice);

    dim3 blk(BLK_SIZ, BLK_SIZ);
    dim3 grid(n / BLK_SIZ, n / BLK_SIZ);
    matmul<<<grid, blk>>>(dev_A, dev_B, dev_D, n);
    // matmul<<<1, 1>>>(dev_A, dev_B, dev_D, n);

    hipMemcpy(&D[0][0], (void *)dev_D, n * n * sizeof(alt_bn128_Fp), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) for (int j = 0; j < n; j++) D[i][j].print_hex();

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_D);

    return 0;
}